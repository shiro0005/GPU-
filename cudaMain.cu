#include "hip/hip_runtime.h"

#define _CRT_SECURE_NO_WARNINGS
#define _USE_MATH_DEFINES

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector> 
#include <algorithm>
#include <random>
#include <cassert>
#include <tuple>
#include <omp.h>
#include <time.h>
#include <math.h>

#define UNREACHABLE() assert(0)
#define Pi 3.14159265358979323846
#define MAX_Sphere 10
int tonemap(double v) {
	return std::min(
		std::max(int(std::pow(v, 1 / 2.2) * 255), 0), 255);

};

//�\����

struct Ray {
	double3 o;
	double3 d;
};

struct Sphere;

struct Hit {
	double t;
	double3 p;
	double3 n;
	const Sphere* sphere;
	bool F;
};

enum class SurfaceType {
	Diffuse,
	Mirror,
	Fresnel,
};

struct Sphere {
	double3 p;
	double r;
	SurfaceType type;
	double3 R;//���˗��@�@�F
	double3 Le;
	double ior = 1.5168;
};

struct Scene {
	Sphere spheres[MAX_Sphere]
	{
		{ double3{1e5 + 1,   40.8,		  81.6}, 1e5 , SurfaceType::Diffuse, double3{.99,0.,0.} },//���̕�
		{ double3{-1e5 + 99, 40.8,		  81.6}, 1e5 , SurfaceType::Diffuse, double3{0.,.99,0.} },//�E�̕�
		{ double3{50,        40.8,		  1e5},  1e5 , SurfaceType::Diffuse, double3{.75,.75,.75} },//���̕�
		{ double3{50,        1e5,		  81.6}, 1e5 , SurfaceType::Diffuse, double3{.75,.75,.75} },//�V��
		{ double3{50,		 -1e5 + 81.6, 81.6}, 1e5 , SurfaceType::Diffuse, double3{.75,.75,.75} },//��
		{ double3{37,		 16.5,		  47},   16.5, SurfaceType::Mirror, double3{.999,.999,.999}  },//�����̋�
		{ double3{37,		 49.5,		  47},   16.5, SurfaceType::Mirror, double3{.999,.999,.999}  },//����̋�
		{ double3{73,		 16.5,		  78},   16.5, SurfaceType::Fresnel,double3{.999,.999,.999}  },//�E���̋�
		{ double3{73,		 49.5,		  78},   16.5, SurfaceType::Fresnel,double3{.999,.999,.999}  },//�E��̋�
		{ double3{50,		 681.6 - .27, 81.6}, 600 , SurfaceType::Diffuse, double3{0,0,0}, double3{12,12,12} },//���C�g
	};
};


//�O���[�o���ϐ�
const int width = 1200;		//�摜��
const int height = 800;		//�摜����



const int spp = 10;			//�s�N�Z�����Ƃ̃T���v����
const int depth = 10;		//���C�̔��ː�


//�z�X�g
double3 h_Result[width*height];

//�f�o�C�X
double3 *d_Result;


//�f�o�C�X�֐�
__device__ double dot(double3 a, double3 b) {
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ double3 Normalize(double3 v) {
	return double3{ v.x / sqrt(dot(v, v)), v.y / sqrt(dot(v, v)), v.z / sqrt(dot(v, v)) };
}


__device__ double3 cross(double3 a, double3 b) {
	return double3{ a.y * b.z - a.z * b.y,
					a.z * b.x - a.x * b.z,
					a.x * b.y - a.y * b.x };
}

//�J�[�l���֐�
// GPU�Ōv�Z����ۂ̊֐�
__global__ void gpu_function(double3 *d_Result)
{
	int k_x = blockIdx.x * blockDim.x + threadIdx.x;//�J�[�l����X���W
	int k_y = blockIdx.y*blockDim.y + threadIdx.y;//�J�[�l����Y���W
	int xsize = gridDim.x*blockDim.x;
	int id = k_x + k_y * xsize;

	d_Result[id] = { 0,0,0 };//{r,g,b}

	hiprandStateXORWOW_t rands;
	hiprand_init(1234, id, 0, &rands);

	/*camera parameter*/

	//�ʒu
	const double3 eye{ 50, 52, 295.6 };
	//�����_
	const double3 center = double3{ eye.x + 0, eye.y - 0.042612, eye.z - 1 };
	//�J�����̏��\���x�N�g��
	const double3 up{ 0, 1, 0 };
	//����p
	const double fov = 30 * Pi / 180;
	//��ʂ̃A�X�y�N�g��
	const double aspect = double(width) / height;


	// Basis vectors for camera coordinates
	//�J�������W�n�̊��x�N�g��
	const auto wE = Normalize({ eye.x - center.x, eye.y - center.y, eye.z - center.z });
	const auto uE = Normalize(cross(up, wE));
	const auto vE = cross(wE, uE);


	for (int j = 0; j < spp; j++) {
		const int x = id % width;
		const int y = height - id / width;
		Ray ray;
		
		ray.o = eye;

		ray.d = [&]() {
			const double tf = tan(fov * .5);
			const double rpx = 2. * (x + hiprand_uniform_double(&rands)) / width - 1;
			const double rpy = 2. * (y + hiprand_uniform_double(&rands)) / height - 1;
			const double3 w = Normalize(double3{ aspect * tf * rpx, tf * rpy, -1 });
			return double3{ uE.x * w.x + vE.x * w.y + wE.x * w.z,
							uE.y * w.x + vE.y * w.y + wE.y * w.z,
							uE.z * w.x + vE.z * w.y + wE.z * w.z };// uE*w.x + vE * w.y + wE * w.z;
		}();

		double3 L{ 0,0,0 };
		double3 th{ 1.,1.,1. };

		for (int k = 0; k < depth; k++) {//���ˉ񐔁H�@�@���ˉ񐔂��P���Ƃ��������C�̐��łQ
			
			// ���_�E�J�����̐ݒ�
			Scene scene;

			Hit minh;
			int num;
			double tmin = 1e-4;
			double tmax = 1e+10;

			for (int i = 0; i < MAX_Sphere; i++) {
				Hit hit;

				const double3 op = { scene.spheres[i].p.x - ray.o.x , scene.spheres[i].p.y - ray.o.y , scene.spheres[i].p.z - ray.o.z };
				
				const double b = op.x*ray.d.x + op.y*ray.d.y + op.z*ray.d.z;

				const double det = b * b - (op.x*op.x + op.y*op.y + op.z*op.z) + scene.spheres[i].r * scene.spheres[i].r;

				if (det < 0) {
					hit = Hit{ 0,{0,0,0},{0,0,0},nullptr,false };
			
				}
				else {
					const double t1 = b - sqrt(det);
					if (tmin < t1 && t1 < tmax) {
					    hit = Hit{ t1, {}, {}, &scene.spheres[i] ,true };
						
					}
					else {
						const double t2 = b + sqrt(det);
						
						if (tmin < t2 && t2 < tmax) {
							num = 11;
							

							hit = Hit{ t2, {}, {}, &scene.spheres[i] ,true };
						}
						else {
						
							hit = Hit{ 0,{0,0,0},{0,0,0},nullptr,false };//�K����HIT�̒l��Ԃ�
						}
					}
				}
													
				if (!hit.F) { continue; };

				//num = i;
				minh = hit;
				minh.F = true;//�ǉ�
				tmax = minh.t;
			}
			
			if (minh.F) {
				const Sphere* s = minh.sphere;
				minh.p = double3{ ray.o.x + ray.d.x * minh.t, ray.o.y + ray.d.y * minh.t, ray.o.z + ray.d.z * minh.t };
				minh.n = double3{ (minh.p.x - s->p.x) / s->r ,(minh.p.y - s->p.y) / s->r ,(minh.p.z - s->p.z) / s->r };
			}
				//return minh;
			


			// Intersection
			const Hit h = minh;
	
			
			if (!h.F) {
				break;
			}

			// Add contribution
			L = double3{ L.x + th.x * h.sphere->Le.x, L.y + th.y * h.sphere->Le.y, L.z + th.z * h.sphere->Le.z };
			
			
			// Update next direction
			ray.o = h.p;
			ray.d = [&]() {
				if (h.sphere->type == SurfaceType::Diffuse) {
					// Sample direction in local coordinates
					const double3 n = dot(h.n, double3{ -ray.d.x,-ray.d.y ,-ray.d.z }) > 0 ? double3{ h.n.x,h.n.y,h.n.z } : double3{ -h.n.x,-h.n.y,-h.n.z };

					double3 u{ 0,0,0 }, v{ 0,0,0 };
					const double s = n.z >= 0 ? 1 : -1;


					const double a = -1 / (s + n.z);
					const double b = n.x * n.y * a;

					u = double3{ 1 + s * n.x * n.x * a, s * b, -s * n.x };
					v = double3{ b, s + n.y * n.y * a, -n.y };


					const double3 d = [&]() {
						const double r = sqrt(hiprand_uniform_double(&rands));
						const double t = 2 * Pi * hiprand_uniform_double(&rands);
						const double x = r * cos(t);
						const double y = r * sin(t);

						if (0.0 > 1 - x * x - y * y) {
							return double3{ x, y,
								sqrt(0.0) };
						}
						else {
							return double3{ x, y,
							sqrt(1 - x * x - y * y) };
						}

					}();

					// Convert to world coordinates
					return  double3{ u.x * d.x + v.x * d.y + n.x * d.z,
									 u.y * d.x + v.y * d.y + n.y * d.z,
									 u.z * d.x + v.z * d.y + n.z * d.z };

				}
				else if (h.sphere->type == SurfaceType::Mirror) {
					
					const double3 wi = double3{ -ray.d.x, -ray.d.y, -ray.d.z };//-ray.d
					return  double3{ 2 * dot(wi,h.n) * h.n.x - wi.x,
									 2 * dot(wi,h.n) * h.n.y - wi.y,
									 2 * dot(wi,h.n) * h.n.z - wi.z };
					
				}
				else if (h.sphere->type == SurfaceType::Fresnel) {

					const double3 wi = double3{ -ray.d.x, -ray.d.y, -ray.d.z };//-ray.d;
					const bool into = dot(wi, h.n) > 0;
					const double3 n = into ? h.n : double3{ -h.n.x,-h.n.y,-h.n.z };
					const double ior = h.sphere->ior;
					const double eta = into ? 1 / ior : ior;

					bool F;
					const double3 wt = [&]() -> double3 {
						// Snell's law (vector form)
						const double t = dot(wi, n);
						const double t2 = 1 - eta * eta * (1 - t * t);

						if (t2 < 0) {
							F = false;
							return double3{ 0,0,0 };
						}

						F = true;
						return double3{ eta * (n.x * t - wi.x) - n.x * sqrt(t2),
										eta * (n.y * t - wi.y) - n.y * sqrt(t2),
										eta * (n.z * t - wi.z) - n.z * sqrt(t2) }; //eta * (n * t - wi) - n * sqrt(t2);

					}();

					if (!F) {
						// Total internal reflection
						return double3{ 2 * dot(wi,h.n) * h.n.x - wi.x,
									    2 * dot(wi,h.n) * h.n.y - wi.y,
									    2 * dot(wi,h.n) * h.n.z - wi.z };// 2 * dot(wi, h.n) * h.n - wi;
					}

					const double Fr = [&]() {
						// Schlick's approximation
						const double cos = into
							? dot(wi, h.n)
							: dot(wt, h.n);
						const double r = (1 - ior) / (1 + ior);
						return r * r + (1 - r * r) * pow(1 - cos, 5);
					}();

					// Select reflection or refraction
					// according to the fresnel term
					return  hiprand_uniform_double(&rands) < Fr
						? double3{ 2 * dot(wi, h.n) * h.n.x - wi.x,
								   2 * dot(wi, h.n) * h.n.y - wi.y,
								   2 * dot(wi, h.n) * h.n.z - wi.z }
					: wt;
				}

				//UNREACHABLE();
				return double3{ 0,0,0 }; 
			}();


			// Update throughput
			th = double3{ th.x*h.sphere->R.x, th.y*h.sphere->R.y ,th.z*h.sphere->R.z };
			if (th.x > th.y&&th.x > th.z&&th.x == 0) {

				break;
			}
			if (th.y > th.x&&th.y > th.z&&th.y == 0) {

				break;
			}
			if (th.z > th.x&&th.z > th.y&&th.z == 0) {

				break;
			}
		}

		d_Result[id] = double3{ (d_Result[id].x + L.x / spp), (d_Result[id].y + L.y / spp), (d_Result[id].z + L.z / spp) };
	}

}

// main function
int cudafunction(void)
{

	int start = clock();

	// �f�o�C�X(GPU)���̗̈�m��
	hipMalloc(&d_Result, width*height * sizeof(double3));
	

	// CPU��GPU�̃f�[�^�R�s�[
	hipMemcpy(d_Result, h_Result, width*height * sizeof(double3), hipMemcpyHostToDevice);
	

	dim3 grid(75, 50);//�O���b�h
	dim3 block(16, 16, 1);//�u���b�N 16�̔{���������炵��



	// GPU�Ōv�Z
	gpu_function << <grid, block >> > (d_Result);

	// GPU��CPU�̃f�[�^�R�s�[
	hipMemcpy(h_Result, d_Result, width*height * sizeof(double3), hipMemcpyDeviceToHost);

	int end = clock();

	hipFree(d_Result);

	std::ofstream ofs("result.ppm");
	ofs << "P3\n" << width << " " << height << "\n255\n";
	for (const auto& i : h_Result) {
		ofs << tonemap(i.x) << " "
			<< tonemap(i.y) << " "
			<< tonemap(i.z) << "\n";
	}

	return end-start;
}

